// naive_matrix_mul.cu
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <chrono>

// CUDA Kernel for naive matrix multiplication
__global__ void matrixMulNaive(float* A, float* B, float* C, int M, int K, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; // Row index of C
    int col = blockIdx.x * blockDim.x + threadIdx.x; // Column index of C

    if (row < M && col < N) {
        float value = 0.0f;
        for (int e = 0; e < K; ++e) {
            value += A[row * K + e] * B[e * N + col];
        }
        C[row * N + col] = value;
    }
}

int main() {
    // Matrix dimensions
    int M = 1024; // Rows of A and C
    int K = 1024; // Columns of A and Rows of B
    int N = 1024; // Columns of B and C

    // Host memory allocation
    std::vector<float> h_A(M * K, 1.0f); // Initialize A with 1.0
    std::vector<float> h_B(K * N, 1.0f); // Initialize B with 1.0
    std::vector<float> h_C(M * N, 0.0f); // Initialize C with 0.0

    // Device memory allocation
    float *d_A, *d_B, *d_C;
    size_t bytes_A = M * K * sizeof(float);
    size_t bytes_B = K * N * sizeof(float);
    size_t bytes_C = M * N * sizeof(float);

    hipMalloc(&d_A, bytes_A);
    hipMalloc(&d_B, bytes_B);
    hipMalloc(&d_C, bytes_C);

    // Copy data from host to device
    hipMemcpy(d_A, h_A.data(), bytes_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), bytes_B, hipMemcpyHostToDevice);

    // Define block and grid sizes
    dim3 block(16, 16);
    dim3 grid((N + block.x - 1) / block.x, (M + block.y - 1) / block.y);

    // Launch kernel
    matrixMulNaive<<<grid, block>>>(d_A, d_B, d_C, M, K, N);

    // Copy result back to host
    hipMemcpy(h_C.data(), d_C, bytes_C, hipMemcpyDeviceToHost);

    // Optional: Verify result
    bool correct = true;
    for (int i = 0; i < M * N; ++i) {
        if (h_C[i] != K * 1.0f) { // Since A and B are initialized to 1.0
            correct = false;
            break;
        }
    }
    if (correct)
        std::cout << "Naive Matrix Multiplication Successful!\n";
    else
        std::cout << "Naive Matrix Multiplication Failed!\n";

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
